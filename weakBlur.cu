#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#define STB_IMAGE_IMPLEMENTATION
#include <stb/stb_image.h>
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include <stb/stb_image_write.h>
#include <iostream>

__global__ void blurImg(unsigned char *d_in, unsigned char *d_out, int w, int h, int ch, int blur, double sum){
    for(unsigned int j = blockIdx.x * blockDim.x + threadIdx.x; j < w; j += gridDim.x * blockDim.x){
        for(unsigned int i = blockIdx.y * blockDim.y + threadIdx.y; i < h; i += gridDim.y * blockDim.y){
            double sumr = 0, sumg = 0, sumb = 0;
            for(int ki = -blur; ki <= blur; ki++){
                for(int kj = -blur; kj <= blur; kj++){
                    int idx = (i + ki) * w * ch + (j + kj) * ch;
                    if(idx >= 0 && idx < w * h * ch){
                        double factor = pow(2, blur * 2);
                        int diff = abs(ki)+abs(kj);
                        factor /= pow(2, diff);
                        factor /= sum;
                        sumr += (double)d_in[idx] * factor;
                        sumg += (double)d_in[idx + 1] * factor;
                        sumb += (double)d_in[idx + 2] * factor;
                    }
                }
            }
            unsigned int idx = i * w * ch + j * ch;
            d_out[idx] = ceil(sumr);
            d_out[idx + 1] = ceil(sumg);
            d_out[idx + 2] = ceil(sumb);
        }
    }
}

int main(int argc, char** argv){
    if(argc != 3){
        std::cout << "Please, provide the correct arguments\n./cudaBlur *blur level 1~9* *image-path*" << std::endl;
        return -1;
    }
    int blursize = argv[1][0] - 48;
    const char* path = argv[2];
    int w, h, ch;
    unsigned char* img = stbi_load(path, &w, &h, &ch, 0);
    unsigned char* res = new unsigned char[w*h*ch];
    unsigned char* d_in, *d_out;
    double V = pow(2, blursize * 2);
    double sum = V;
    double marginal = 1;
    double qtd = 0;
    for(int i = 1; i <= blursize; i++){
        qtd += 4;
        sum += V * qtd / pow(2, i);
        sum += marginal * qtd;
        marginal *= 2;
    }
    dim3 bNum(16, 16);
    dim3 tNum(16, 16);
    hipMalloc((void**) &d_in, sizeof(unsigned char) * w * h * ch);
    hipMalloc((void**) &d_out, sizeof(unsigned char) * w * h * ch);
    hipMemcpy(d_in, img, sizeof(unsigned char) * w * h * ch, hipMemcpyHostToDevice);
    blurImg<<<bNum, tNum>>>(d_in, d_out, w, h, ch, blursize, sum);
    hipDeviceSynchronize();
    hipMemcpy(res, d_out, sizeof(unsigned char) * w * h * ch, hipMemcpyDeviceToHost);
    stbi_write_jpg("../images/output.jpg", w, h, ch, res, 100);
    hipFree(d_in); hipFree(d_out); delete res; stbi_image_free(img);
}